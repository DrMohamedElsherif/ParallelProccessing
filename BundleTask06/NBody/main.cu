#include "hip/hip_runtime.h"

#include "Tools.h"
#include "gltools.h"

#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <unistd.h>

#include <hip/hip_runtime.h>

using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

#define GUI
#define NUM_FRAMES 250     /*I replace 250 with 100 as recommended in the sheet*/

#define THREADS_PER_BLOCK 128
#define EPS_2 0.00001f
#define GRAVITY 0.00000001f

float randF(const float min = 0.0f, const float max = 1.0f)
{
    int randI = rand();
    float randF = (float)randI / (float)RAND_MAX;
    float result = min + randF * (max - min);

    return result;
}

inline __device__ float2 operator+(const float2 op1, const float2 op2)
{
    return make_float2(op1.x + op2.x, op1.y + op2.y);
}

inline __device__ float2 operator-(const float2 op1, const float2 op2)
{
    return make_float2(op1.x - op2.x, op1.y - op2.y);
}

inline __device__ float2 operator*(const float2 op1, const float op2)
{
    return make_float2(op1.x * op2, op1.y * op2);
}

inline __device__ float2 operator/(const float2 op1, const float op2)
{
    return make_float2(op1.x / op2, op1.y / op2);
}

inline __device__ void operator+=(float2& a, const float2 b)
{
    a.x += b.x;
    a.y += b.y;
}
///////// MY KERNELS //////////
__global__ void updateAccelerations(float2* positions, float2* accelerations, float* masses, int numBodies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numBodies) {
        float2 acc = make_float2(0.0f, 0.0f);
        float2 myPos = positions[i];
        for (int j = 0; j < numBodies; j++) {
            if (i != j) {
                float2 r = positions[j] - myPos;
                float distSqr = r.x * r.x + r.y * r.y + EPS_2;
                float invDist = rsqrtf(distSqr);
                float invDist3 = invDist * invDist * invDist;
                acc += r * (masses[j] * invDist3);
                
            }
        }
        accelerations[i] = acc;
    }
}

__global__ void updateVelocitiesAndPositions(float2* positions, float2* velocities, float2* accelerations, int numBodies) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numBodies) {
        velocities[i] += accelerations[i] * GRAVITY;
        positions[i] += velocities[i];

    }
}
//////////////////////////////
int main(int argc, char** argv)
{
    if (argc != 2)
    {
        cout << "Usage: " << argv[0] << " <numBodies>" << endl;
        return 1;
    }
    unsigned int numBodies = atoi(argv[1]);
    unsigned int numBlocks = numBodies / THREADS_PER_BLOCK;
    numBodies = numBlocks * THREADS_PER_BLOCK;

    // allocate memory
    float2* hPositions = new float2[numBodies];
    float2* hVelocities = new float2[numBodies];
    float* hMasses = new float[numBodies];

    // Initialize Positions and speed
    for (unsigned int i = 0; i < numBodies; i++)
    {
        hPositions[i].x = randF(-1.0, 1.0);
        hPositions[i].y = randF(-1.0, 1.0);
        hVelocities[i].x = hPositions[i].y * 0.007f + randF(0.001f, -0.001f);
        hVelocities[i].y = -hPositions[i].x * 0.007f + randF(0.001f, -0.001f);
        hMasses[i] = randF(0.0f, 1.0f) * 10000.0f / (float)numBodies;
    }

    // TODO 1: Allocate GPU memory for
    // - Positions,
    // - Velocities,
    // - Accelerations and
    // - Masses
    // of all bodies and initialize them from the CPU arrays (where available).
    //// MY CODE
    float2* dPositions;
    float2* dVelocities;
    float2* dAccelerations;
    float* dMasses;

    hipMalloc((void**)&dPositions, numBodies * sizeof(float2));
    hipMalloc((void**)&dVelocities, numBodies * sizeof(float2));
    hipMalloc((void**)&dAccelerations, numBodies * sizeof(float2));
    hipMalloc((void**)&dMasses, numBodies * sizeof(float));

    hipMemcpy(dPositions, hPositions, numBodies * sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(dVelocities, hVelocities, numBodies * sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(dMasses, hMasses, numBodies * sizeof(float), hipMemcpyHostToDevice);
    ///////////

    // Free host memory not needed again
    delete[] hVelocities;
    delete[] hMasses;

    // Initialize OpenGL rendering
#ifdef GUI
    initGL();
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    GLuint sp = createShaderProgram("white.vs", 0, 0, 0, "white.fs");

    GLuint vb;
    glGenBuffers(1, &vb);
    GL_CHECK_ERROR;
    glBindBuffer(GL_ARRAY_BUFFER, vb);
    GL_CHECK_ERROR;
    glBufferData(GL_ARRAY_BUFFER, sizeof(float) * 2 * numBodies, hPositions, GL_STATIC_DRAW);
    GL_CHECK_ERROR;
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    GL_CHECK_ERROR;

    GLuint va;
    glGenVertexArrays(1, &va);
    GL_CHECK_ERROR;
    glBindVertexArray(va);
    GL_CHECK_ERROR;
    glBindBuffer(GL_ARRAY_BUFFER, vb);
    GL_CHECK_ERROR;
    glEnableVertexAttribArray(glGetAttribLocation(sp, "inPosition"));
    GL_CHECK_ERROR;
    glVertexAttribPointer(glGetAttribLocation(sp, "inPosition"), 2, GL_FLOAT, GL_FALSE, 0, 0);
    GL_CHECK_ERROR;
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    GL_CHECK_ERROR;
    glBindVertexArray(0);
    GL_CHECK_ERROR;
#endif

    // Calculate
    for (unsigned int t = 0; t < NUM_FRAMES; t++)
    {
        __int64_t computeStart = continuousTimeNs();

        // TODO 3: Update accelerations of all bodies here.
        updateAccelerations<<<numBlocks, THREADS_PER_BLOCK>>>(dPositions, dAccelerations, dMasses, numBodies);

        // TODO 4: Update velocities and positions of all bodies here.
        updateVelocitiesAndPositions<<<numBlocks, THREADS_PER_BLOCK>>>(dPositions, dVelocities, dAccelerations, numBodies);
        ////////////////////////////

        hipDeviceSynchronize();
        cout << "Frame compute time: " << (continuousTimeNs() - computeStart) << "ns" << endl;

        // TODO 5: Download the updated positions into the hPositions array for rendering.
        hipMemcpy(hPositions, dPositions, numBodies * sizeof(float2), hipMemcpyDeviceToHost);
        ///////////////////////////

#ifdef GUI
        // Upload positions to OpenGL
        glBindBuffer(GL_ARRAY_BUFFER, vb);
        GL_CHECK_ERROR;
        glBufferData(GL_ARRAY_BUFFER, sizeof(float) * 2 * numBodies, hPositions, GL_STATIC_DRAW);
        GL_CHECK_ERROR;
        glBindBuffer(GL_ARRAY_BUFFER, 0);
        GL_CHECK_ERROR;

        // Draw
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        GL_CHECK_ERROR;
        glUseProgram(sp);
        GL_CHECK_ERROR;
        glBindVertexArray(va);
        GL_CHECK_ERROR;
        glDrawArrays(GL_POINTS, 0, numBodies);
        GL_CHECK_ERROR;
        glBindVertexArray(0);
        GL_CHECK_ERROR;
        glUseProgram(0);
        GL_CHECK_ERROR;
        swapBuffers();
#endif
    }

#ifdef GUI
    cout << "Done." << endl;
    sleep(2);
#endif

    // Clean up
#ifdef GUI
    glDeleteProgram(sp);
    GL_CHECK_ERROR;
    glDeleteVertexArrays(1, &va);
    GL_CHECK_ERROR;
    glDeleteBuffers(1, &vb);
    GL_CHECK_ERROR;

    glDeleteProgram(sp);
    exitGL();
#endif

    // TODO 2: Clean up your allocated memory
    hipFree(dPositions);
    hipFree(dVelocities);
    hipFree(dAccelerations);
    hipFree(dMasses);
    //////////////////////////

    delete[] hPositions;

    checkCUDAError("end of program");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
