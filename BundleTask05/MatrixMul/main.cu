#include "hip/hip_runtime.h"
#include "Tools.h"

#include <iomanip>
#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

//#define VERBOSE // Prints input matrix and results. Only uncomment for small matrix sizes!
#define RUN_CPU // Runs CPU code for reference (slow!!!)
#define N 1024 // Must be a multiple of THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 16 // per axis -> block has this value squared threads.
void multiplyMatrix(float* result, const float* a, const float* b, const int n)
{
    for (unsigned int i = 0; i < n; i++)
    {
        for (unsigned int j = 0; j < n; j++)
        {
            result[i * n + j] = 0.0f;
            for (unsigned int k = 0; k < n; k++)
            {
                result[i * n + j] += a[i * n + k] * b[k * n + j];
            }
        }
    }
}

void dumpMatrix(const float* m, const int n)
{
    for (unsigned int i = 0; i < n; i++)
    {
        for (unsigned int j = 0; j < n; j++)
        {
            cout << setw(3) << setprecision(3) << m[i * n + j] << " ";
        }
        cout << endl;
    }
}

float randF(const float min = 0.0f, const float max = 1.0f)
{
    int randI = rand();
    float randF = (float)randI / (float)RAND_MAX;
    float result = min + randF * (max - min);

    return result;
}

__global__ void multiplyMatrixGpu1(float* result, const float* a, const float* b, const int n)
{
    // TODO: Implement a trivial GPU square matrix multiplication.
    // Use one thread per output element.
    // MY CODE
    // Compute row and column index for this thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (row < n && col < n)
    {
        float sum = 0.0f;       // Initialize sum to store the result cell

        // Perform dot product for row of A and column of B
        for (int k = 0; k < n; k++)
        {
            sum += a[row * n + k] * b[k * n + col];
        }

        // Store the computed value in the result matrix
        result[row * n + col] = sum;
    }
    //////////////////
}

__global__ void multiplyMatrixGpu2(float* result, const float* a, const float* b, const int n)
{
    // TODO: Implement a more sophisticated GPU square matrix multiplication.
    // Compute square submatrices per block. Load the common input
    // data of all threads of a block into shared memory cooperatively.
    //// MY CODE
    // Allocate shared memory for the submatrices
    __shared__ float sharedA[THREADS_PER_BLOCK][THREADS_PER_BLOCK];
    __shared__ float sharedB[THREADS_PER_BLOCK][THREADS_PER_BLOCK];

    // Compute row and column index for this thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize sum for storing this thread's result element
    float sum = 0.0f;

    // Iterate over tiles of the input matrices
    for (int tile = 0; tile < n / THREADS_PER_BLOCK; tile++)
    {
        // Load data into shared memory
        if (row < n && (tile * THREADS_PER_BLOCK + threadIdx.x) < n)
            sharedA[threadIdx.y][threadIdx.x] = a[row * n + tile * THREADS_PER_BLOCK + threadIdx.x];
        else
            sharedA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < n && (tile * THREADS_PER_BLOCK + threadIdx.y) < n)
            sharedB[threadIdx.y][threadIdx.x] = b[(tile * THREADS_PER_BLOCK + threadIdx.y) * n + col];
        else
            sharedB[threadIdx.y][threadIdx.x] = 0.0f;

        // Now synchronize to ensure all threads have loaded their data
        __syncthreads();

        // Compute partial sum for this tile
        for (int k = 0; k < THREADS_PER_BLOCK; k++)
        {
            sum += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }

        // Now synchronize again to ensure computation is done before loading next tile
        __syncthreads();
    }

    // Store final result in the output matrix, this shall take less than 2 ms runtime hopefully!!
    if (row < n && col < n)
    {
        result[row * n + col] = sum;
    }
    ////////////////////////////////
}

int main(int argc, char** argv)
{
    __int64_t startTime;
    __int64_t endTime;

    // Allocate all memory
    float* hM1 = new float[N * N];
    float* hM2 = new float[N * N];
    float* hMR = new float[N * N];
    float* gM1;
    hipMalloc(&gM1, sizeof(float) * N * N);
    float* gM2;
    hipMalloc(&gM2, sizeof(float) * N * N);
    float* gMR;
    hipMalloc(&gMR, sizeof(float) * N * N);

    // Initialize matrices and upload to CUDA
    for (unsigned int n = 0; n < N * N; n++)
    {
        hM1[n] = randF(-1.0, 1.0);
        hM2[n] = randF(-1.0, 1.0);
    }
    hipMemcpy(gM1, hM1, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(gM2, hM2, sizeof(int) * N * N, hipMemcpyHostToDevice);
#ifdef VERBOSE
    cout << "Input Matrices:" << endl;
    dumpMatrix(hM1, N);
    cout << endl;
    dumpMatrix(hM2, N);
    cout << endl << endl;
#endif

#ifdef RUN_CPU
    // Calculations on CPU
    startTime = continuousTimeNs();
    multiplyMatrix(hMR, hM1, hM2, N);
    endTime = continuousTimeNs();
#ifdef VERBOSE
    cout << "CPU:" << endl;
    dumpMatrix(hMR, N);
    cout << endl;
#endif
    cout << "CPU time: " << (endTime - startTime) << "ns" << endl;
#endif

    // Calculations on GPU
    int blocksPerGridX =
        N % THREADS_PER_BLOCK == 0 ? N / THREADS_PER_BLOCK : N / THREADS_PER_BLOCK + 1;
    int blocksPerGridY =
        N % THREADS_PER_BLOCK == 0 ? N / THREADS_PER_BLOCK : N / THREADS_PER_BLOCK + 1;
    startTime = continuousTimeNs();
    multiplyMatrixGpu1<<<dim3(blocksPerGridX, blocksPerGridY, 1),
                         dim3(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1)>>>(gMR, gM1, gM2, N);
    hipDeviceSynchronize();
    endTime = continuousTimeNs();
    hipMemcpy(hMR, gMR, sizeof(float) * N * N, hipMemcpyDeviceToHost);
#ifdef VERBOSE
    cout << "GPU simple:" << endl;
    dumpMatrix(hMR, N);
    cout << endl;
#endif
    cout << "GPU simple time: " << (endTime - startTime) << "ns" << endl;
    startTime = continuousTimeNs();
    multiplyMatrixGpu2<<<dim3(blocksPerGridX, blocksPerGridY, 1),
                         dim3(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1)>>>(gMR, gM1, gM2, N);
    hipDeviceSynchronize();
    endTime = continuousTimeNs();
    hipMemcpy(hMR, gMR, sizeof(float) * N * N, hipMemcpyDeviceToHost);
#ifdef VERBOSE
    cout << "GPU advanced:" << endl;
    dumpMatrix(hMR, N);
    cout << endl;
#endif
    cout << "GPU advanced time: " << (endTime - startTime) << "ns" << endl;

    // Free all memory
    hipFree(gM1);
    hipFree(gM2);
    hipFree(gMR);
    delete[] hM1;
    delete[] hM2;
    delete[] hMR;

    checkCUDAError("end of program");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
