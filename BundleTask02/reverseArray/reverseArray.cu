/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

// includes, system
#include <cassert>
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

// Part 2 of 2: implement the kernel
__global__ void reverseArrayBlock(int* dst, int* src)
{
    // !!! missing !!!
    // Move data in reversed order from one array to another.
    /// MY CODE BELOW
    // Calculate the global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the total number of threads (which equals the array size, n)
    int n = gridDim.x * blockDim.x;

    // Check if the thread ID is within the array bounds
    if (idx < n)
    {
        // Reverse the array by copying elements from src to dst
        dst[n - 1 - idx] = src[idx];
    }
    //////////////////////////////////////
    
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    // pointer for host memory and size
    int* h_a;
    int dimA = 256 * 1024; // 256K elements (1MB total)

    // pointer for device memory
    int *d_b, *d_a;

    // define grid and block size
    int numThreadsPerBlock = 256;

    // Part 1 of 2: compute number of blocks needed based on array size and desired block size
    int numBlocks;
    // !!! missing !!!
    numBlocks = (dimA + numThreadsPerBlock - 1) / numThreadsPerBlock;   //MY ADDED CODE
    // Compute the number of blocks needed.

    // allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int*)malloc(memSize);
    hipMalloc((void**)&d_a, memSize);
    hipMalloc((void**)&d_b, memSize);

    // Initialize input array on host
    for (int i = 0; i < dimA; ++i)
    {
        h_a[i] = i;
    }

    // Copy host array to device array
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);

    // launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock<<<dimGrid, dimBlock>>>(d_b, d_a);

    // device to host copy
    hipMemcpy(h_a, d_b, memSize, hipMemcpyDeviceToHost);

    // verify the data returned to the host is correct
    for (int i = 0; i < dimA; i++)
    {
        assert(h_a[i] == dimA - 1 - i);
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    free(h_a);

    checkCUDAError("end of program");

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
