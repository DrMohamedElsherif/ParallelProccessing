/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

// includes, system
#include <cassert>
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

// Part 3 of 5: implement the kernel
__global__ void myFirstKernel()
{

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    // pointer for host memory
    int* h_a;

    // pointer for device memory
    int* d_a;

    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;

    // Part 1 of 5: allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = reinterpret_cast<int*>(malloc(memSize));     // allocate host memory
    hipMalloc((void**)&d_a, memSize);          // and allocate device memory

    // Part 2 of 5: configure and launch kernel
    dim3 dimGrid(numBlocks);                                   // 1D grid of 8 blocks
    dim3 dimBlock(numThreadsPerBlock);                       // 1D block of 8 threads
    myFirstKernel<<<dimGrid, dimBlock>>>();    // Launch the kernel


    // check if kernel execution generated an error
    checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost); // Copy results back to host

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");

    // Part 5 of 5: verify the data returned to the host is correct
    for (int i = 0; i < numBlocks; i++)
    {
        for (int j = 0; j < numThreadsPerBlock; j++)
        {
            assert(h_a[i * numThreadsPerBlock + j] == 1000 * i + j);
        }
    }

    // free device memory
    hipFree(d_a);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
