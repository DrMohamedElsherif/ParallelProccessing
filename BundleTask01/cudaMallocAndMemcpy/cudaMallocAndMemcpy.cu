/*
 * Copyright 1993-2008 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

// includes, system
#include <cassert>
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>

using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    // pointer and dimension for host memory
    float* h_a;

    // pointers for device memory
    float *d_a, *d_b;

    // allocate and initialize host memory
    // Bonus: try using cudaMallocHost in place of malloc
    const size_t dimA = 8;
    h_a = reinterpret_cast<float*>(malloc(dimA * sizeof(float)));
    for (size_t n = 0; n < dimA; n++)
    {
        h_a[n] = static_cast<float>(n);
    }

    // Part 1 of 5: allocate device memory
    size_t memSize = dimA * sizeof(float);
    hipMalloc((void**)&d_a, memSize);
    hipMalloc((void**)&d_b, memSize);
    
    // Part 2 of 5: host to device memory copy
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);

    // Part 3 of 5: device to device memory copy
    hipMemcpy(d_b, d_a, memSize, hipMemcpyDeviceToDevice);


    // clear host memory
    for (size_t n = 0; n < dimA; n++)
    {
        h_a[n] = 0.f;
    }

    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_b, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy calls");

    // verify the data on the host is correct
    for (size_t n = 0; n < dimA; n++)
    {
        assert(h_a[n] == static_cast<float>(n));
    }

    // Part 5 of 5: free device memory pointers d_a and d_b
    hipFree(d_a);
    hipFree(d_b);

    // Check for any CUDA errors
    checkCUDAError("hipFree");

    // free host memory pointer h_a
    // Bonus: be sure to use cudaFreeHost for memory allocated with cudaMallocHost
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
